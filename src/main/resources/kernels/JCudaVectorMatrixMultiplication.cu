
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiplication(char* M, char* N, char* P, int Width)
{
    int tid, tx, ty;
	tx = blockDim.x*blockIdx.x + threadIdx.x;
	ty = blockDim.y*blockIdx.y + threadIdx.y;
	tid = Width*ty + tx;
       
   	char Value = 0;
	char MVal = 0;
	char NVal = 0;

	for (int i = 0; i < Width; i++)
	{
		MVal = M[ty * Width + i];
		NVal = N[i * Width + tx];
		Value += MVal * NVal;
	}

	P[tid] = Value;

}